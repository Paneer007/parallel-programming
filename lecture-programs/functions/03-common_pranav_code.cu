#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 32

__host__ __device__ void fun(int *arr, int nn)
{
    for (unsigned ii = 0; ii < nn; ++ii)
        ++arr[ii];
}
__global__ void dfun(int *arr)
{
    fun(arr + threadIdx.x, 1);
    // need to change for more blocks.
}
int main()
{
    int arr[N], *darr;
    hipMalloc(&darr, N * sizeof(int));
    for (unsigned ii = 0; ii < N; ++ii)
        arr[ii] = ii;
    hipMemcpy(darr, arr, N * sizeof(int),
               hipMemcpyHostToDevice);
    fun(arr, N);
    dfun<<<1, N>>>(darr);
    hipDeviceSynchronize();
    return 0;
}