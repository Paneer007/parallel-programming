#include <iostream>
#include <hip/hip_runtime.h>

struct nodeAOS {
    int a;
    double b;
    char c;
} *allnodesAOS;

struct nodeSOA {
    int *a;
    double *b;
    char *c;
} allnodesSOA;

__global__ void dkernelaos(nodeAOS *allnodesAOS) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    allnodesAOS[id].a = id;
    allnodesAOS[id].b = 0.0;
    allnodesAOS[id].c = 'c';
}

__global__ void dkernelsoa(int *a, double *b, char *c) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    a[id] = id;
    b[id] = 0.0;
    c[id] = 'd';
}

int main() {
    const int N = 10; // Define the number of elements
    const size_t size = N * sizeof(int);

    // Allocate host memory for AOS
    nodeAOS *h_allnodesAOS = new nodeAOS[N];

    // Allocate host memory for SOA
    nodeSOA h_allnodesSOA;
    h_allnodesSOA.a = new int[N];
    h_allnodesSOA.b = new double[N];
    h_allnodesSOA.c = new char[N];

    // Allocate device memory for AOS
    nodeAOS *d_allnodesAOS;
    hipMalloc(&d_allnodesAOS, N * sizeof(nodeAOS));

    // Allocate device memory for SOA
    int *d_a;
    double *d_b;
    char *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_c, N * sizeof(char));

    dkernelaos<<<N, 1>>>(d_allnodesAOS);
    hipDeviceSynchronize();

    dkernelsoa<<<N, 1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    // Copy data from device to host
    hipMemcpy(h_allnodesAOS, d_allnodesAOS, N * sizeof(nodeAOS), hipMemcpyDeviceToHost);
    hipMemcpy(h_allnodesSOA.a, d_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_allnodesSOA.b, d_b, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_allnodesSOA.c, d_c, N * sizeof(char), hipMemcpyDeviceToHost);

    // Print results for AOS
    std::cout << "Array of Structures (AOS) data:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << "Node " << i << ": a = " << h_allnodesAOS[i].a << ", b = " << h_allnodesAOS[i].b << ", c = " << h_allnodesAOS[i].c << std::endl;
    }

    // Print results for SOA
    std::cout << "Structure of Arrays (SOA) data:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << "Node " << i << ": a = " << h_allnodesSOA.a[i] << ", b = " << h_allnodesSOA.b[i] << ", c = " << h_allnodesSOA.c[i] << std::endl;
    }

    // Free memory
    delete[] h_allnodesAOS;
    delete[] h_allnodesSOA.a;
    delete[] h_allnodesSOA.b;
    delete[] h_allnodesSOA.c;
    hipFree(d_allnodesAOS);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
