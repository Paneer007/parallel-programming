#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
    if (threadIdx.x == 0 ) {
            printf("%d %d %d %d %d %d.\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
        }
}

int main() {
    dim3 grid(2, 3, 4);
    dim3 block(5, 6, 7);
    dkernel<<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}