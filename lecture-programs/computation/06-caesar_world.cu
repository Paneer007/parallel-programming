#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void dkernel(char *arr, int arrlen) {
 unsigned id = threadIdx.x;
 // Note we handle end of array index
 if (id < arrlen) {
    ++arr[id];
 }
}

int main(){
    char cpuarr[] = "Gdkkn\x1fVnqkc-", *gpuarr;

    hipMalloc(&gpuarr, sizeof(char) * (1 + strlen(cpuarr)));
    hipMemcpy(gpuarr, cpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyHostToDevice);
    dkernel<<<1, 32>>>(gpuarr, strlen(cpuarr));
    hipDeviceSynchronize(); // unnecessary, but okay.
    hipMemcpy(cpuarr, gpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyDeviceToHost);
    printf(cpuarr);
}

/*
Output:
Hello World
*/