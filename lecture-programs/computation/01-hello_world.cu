#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
 printf("Hello World.\n");
}

int main() {
 dkernel<<<1, 1>>>();
 dkernel<<<1, 1>>>();
 dkernel<<<1, 1>>>();
 hipDeviceSynchronize();
 return 0;
}

/*
Output:
Hello World.
Hello World.
Hello World.
*/