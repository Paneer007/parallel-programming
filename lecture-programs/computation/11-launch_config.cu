#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(unsigned *vector, unsigned vectorsize)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    // Handles out of bound addressing
    if (id < vectorsize)
        vector[id] = id;
}

#define BLOCKSIZE 1024

int main(int nn, char *str[])
{
    unsigned N = atoi(str[1]);
    unsigned *vector, *hvector;
    hipMalloc(&vector, N * sizeof(unsigned));
    hvector = (unsigned *)malloc(N * sizeof(unsigned));
    // Floating point division
    unsigned nblocks = ceil((float)N / BLOCKSIZE);
    printf("nblocks = %d\n", nblocks);
    dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
    hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
    for (unsigned ii = 0; ii < N; ++ii)
    {
        printf("%4d ", hvector[ii]);
    }
    return 0;
}